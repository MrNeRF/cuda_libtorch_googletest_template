#include <hip/hip_runtime.h>
#include "error_checking.cuh"
#include <iostream>

int main(int argc, char** argv) {
    float* d_input;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_input, 784 * sizeof(float))); // Allocate device memory for input
    hipFree(d_input);
    std::cout << "Hello World!\n";
    return 0;
}