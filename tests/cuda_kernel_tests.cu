#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <iostream>
#include <random>
#include <torch/torch.h>
#include <vector>

struct Libtorch_Simple_Net : torch::nn::Module {
    Libtorch_Simple_Net(int inputDim1, int outputDim1, int inputDim2, int outputDim2, int inputDim3, int outputDim3) {
        // Construct and register two Linear submodules.
        fc1 = register_module("fc1", torch::nn::Linear(inputDim1, outputDim1));
        fc2 = register_module("fc2", torch::nn::Linear(inputDim2, outputDim2));
        fc3 = register_module("fc3", torch::nn::Linear(inputDim3, outputDim3));
    }

    // Implement the Net's algorithm.
    std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> forward(torch::Tensor x) {
        torch::Tensor x1, x2, x3;

        x1 = torch::relu(fc1->forward(x));
        x2 = torch::relu(fc2->forward(x1));
        x3 = fc3->forward(x2);

        return std::make_tuple(x1, x2, x3);
    }

    // Use one of many "standard library" modules.
    torch::nn::Linear fc1{nullptr}, fc2{nullptr}, fc3{nullptr};
};

TEST(ForwardPassLossLibtorch, BasicTest) {
    std::vector<int> labels{0, 1, 0, 1, 0};
    std::vector<float> input{0.1f, 0.1f, 0.1f, 0.1f,
                             0.2f, 0.2f, 0.2f, 0.2f,
                             0.3f, 0.3f, 0.3f, 0.3f,
                             0.4f, 0.4f, 0.4f, 0.4f,
                             0.5f, 0.5f, 0.5f, 0.5f};

    float h_weights1[12] = {-.1f, .2f, .2f, .2f,
                            .5f, -.6f, .7f, .8f,
                            .9f, .10f, -.11f, .12f};
    float h_biases1[3] = {.1f, .2f, .3f};

    float h_weights2[9] = {-.1f, .2f, .3f,
                           .4f, .5f, -.6f,
                           .7f, .8f, .9f};
    float h_biases2[3] = {.4f, .2f, .3f};

    float h_weights3[6] = {-.1f, .2f, .3f,
                           .4f, .5f, -.6f};
    float h_biases3[2] = {.1f, .2f};

    // Prepare device memory
    auto options = torch::TensorOptions().dtype(torch::kFloat32);
    auto inputTensor = torch::from_blob(input.data(), {5, 4}, options).requires_grad_(true);
    auto weightTensor1 = torch::from_blob(h_weights1, {3, 4}, options).requires_grad_(true);
    auto biasTensor1 = torch::from_blob(h_biases1, {3}, options).requires_grad_(true);
    auto weightTensor2 = torch::from_blob(h_weights2, {3, 3}, options).requires_grad_(true);
    auto biasTensor2 = torch::from_blob(h_biases2, {3}, options).requires_grad_(true);
    auto weightTensor3 = torch::from_blob(h_weights3, {2, 3}, options).requires_grad_(true);
    auto biasTensor3 = torch::from_blob(h_biases3, {2}, options).requires_grad_(true);

    auto torchNet = std::make_shared<Libtorch_Simple_Net>(4, 3,
                                                          3, 3,
                                                          3, 2);

    torchNet->fc1->weight = weightTensor1;
    torchNet->fc1->bias = biasTensor1;
    torchNet->fc2->weight = weightTensor2;
    torchNet->fc2->bias = biasTensor2;
    torchNet->fc3->weight = weightTensor3;
    torchNet->fc3->bias = biasTensor3;

    auto [pred1, pred2, pred3] = torchNet->forward(inputTensor);

    // This step is necessary because torch is picky about the input type
    // and the labels are int64_t. If you insert std::vector<int>, the loss computation explodes.
    std::vector<int64_t> labels_torch_long(labels.begin(), labels.end());

    auto tensorLables = torch::from_blob(labels_torch_long.data(), {5}, torch::TensorOptions().dtype(torch::kLong));
    auto libtorch_loss = torch::nn::functional::cross_entropy(pred3, tensorLables);
    EXPECT_NEAR(0.70234048366546631f, libtorch_loss.item<float>(), 1e-4);

    torch::optim::SGD optimizer(torchNet->parameters(), /*lr=*/0.01);
    optimizer.zero_grad();
    libtorch_loss.backward();
    optimizer.step();
}

int main(int argc, char** argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
